#include "hip/hip_runtime.h"
/* Copyright (C) 1883 Thomas Edison - All Rights Reserved
 * You may use, distribute and modify this code under the
 * terms of the GPLv3 license, which unfortunately won't be
 * written for another century.
 *
 * You should have received a copy of the LICENSE file with
 * this file.
 */

#include "ethash_cuda_miner_kernel.h"

#include "ethash_cuda_miner_kernel_globals.h"

#include "cuda_helper.h"

#include "fnv.cuh"

#define copy(dst, src, count)                                                                                          \
    for (int i = 0; i != count; ++i) {                                                                                 \
        (dst)[i] = (src)[i];                                                                                           \
    }

#include "keccak.cuh"

#include "dagger_shuffled.cuh"
int RandIndex = 0;

__global__ void ethash_search(Search_results* g_output, uint64_t start_nonce, int kernel) {
    if (g_output->done)
        return;
    uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
    bool r = compute_hash(start_nonce + gid, kernel);
    if (threadIdx.x == 0)
        atomicInc((uint32_t*)&g_output->hashCount, 0xffffffff);
    if (r)
        return;
    uint32_t index = atomicInc((uint32_t*)&g_output->solCount, 0xffffffff);
    if (index >= MAX_SEARCH_RESULTS)
        return;
    g_output->gid[index] = gid;
    g_output->done = 1;
}

void run_ethash_search(uint32_t gridSize, uint32_t blockSize, hipStream_t stream, Search_results* g_output,
                       uint64_t start_nonce, int k) {

    if(k == 0) {
        const int arrayNum[8] = {1, 2, 3, 4, 5, 6, 7, 8};
        k = arrayNum[RandIndex];            
    }    
    ethash_search<<<gridSize, blockSize, 0, stream>>>(g_output, start_nonce, k);
    if(k == 0) (RandIndex >= 8) ? RandIndex = 0 : RandIndex++;

    CUDA_CALL(hipGetLastError());
}

#define ETHASH_DATASET_PARENTS 256
#define NODE_WORDS (64 / 4)

__global__ void ethash_calculate_dag_item(uint32_t start) {
    uint32_t const node_index = start + blockIdx.x * blockDim.x + threadIdx.x;
    if (((node_index >> 1) & (~1)) >= d_dag_size)
        return;
    union {
        hash128_t dag_node;
        uint2 sha3_buf[25];
    };
    copy(dag_node.uint4s, d_light[node_index % d_light_size].uint4s, 4);
    dag_node.words[0] ^= node_index;
    SHA3_512(sha3_buf);

    const int thread_id = threadIdx.x & 3;

    for (uint32_t i = 0; i != ETHASH_DATASET_PARENTS; ++i) {
        uint32_t parent_index = fnv(node_index ^ i, dag_node.words[i % NODE_WORDS]) % d_light_size;
        for (uint32_t t = 0; t < 4; t++) {
            uint32_t shuffle_index = SHFL(parent_index, t, 4);

            uint4 p4 = d_light[shuffle_index].uint4s[thread_id];
            for (int w = 0; w < 4; w++) {
                uint4 s4 = make_uint4(SHFL(p4.x, w, 4), SHFL(p4.y, w, 4), SHFL(p4.z, w, 4), SHFL(p4.w, w, 4));
                if (t == thread_id) {
                    dag_node.uint4s[w] = fnv4(dag_node.uint4s[w], s4);
                }
            }
        }
    }
    SHA3_512(sha3_buf);
    hash64_t* dag_nodes = (hash64_t*)d_dag;
    copy(dag_nodes[node_index].uint4s, dag_node.uint4s, 4);
}

void ethash_generate_dag(uint64_t dag_size, uint32_t gridSize, uint32_t blockSize, hipStream_t stream) {
    const uint32_t work = (uint32_t)(dag_size / sizeof(hash64_t));
    const uint32_t run = gridSize * blockSize;

    uint32_t base;
    for (base = 0; base <= work - run; base += run) {
        ethash_calculate_dag_item<<<gridSize, blockSize, 0, stream>>>(base);
        CUDA_CALL(hipDeviceSynchronize());
    }
    if (base < work) {
        uint32_t lastGrid = work - base;
        lastGrid = (lastGrid + blockSize - 1) / blockSize;
        ethash_calculate_dag_item<<<lastGrid, blockSize, 0, stream>>>(base);
        CUDA_CALL(hipDeviceSynchronize());
    }
    CUDA_CALL(hipGetLastError());
}

void set_constants(hash128_t* _dag, uint32_t _dag_size, hash64_t* _light, uint32_t _light_size) {
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_dag), &_dag, sizeof(hash128_t*)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), &_dag_size, sizeof(uint32_t)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_light), &_light, sizeof(hash64_t*)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_light_size), &_light_size, sizeof(uint32_t)));
}

void get_constants(hash128_t** _dag, uint32_t* _dag_size, hash64_t** _light, uint32_t* _light_size) {
    /*
       Using the direct address of the targets did not work.
       So I've to read first into local variables when using hipMemcpyFromSymbol()
    */
    if (_dag) {
        hash128_t* _d;
        CUDA_CALL(hipMemcpyFromSymbol(&_d, HIP_SYMBOL(d_dag), sizeof(hash128_t*)));
        *_dag = _d;
    }
    if (_dag_size) {
        uint32_t _ds;
        CUDA_CALL(hipMemcpyFromSymbol(&_ds, HIP_SYMBOL(d_dag_size), sizeof(uint32_t)));
        *_dag_size = _ds;
    }
    if (_light) {
        hash64_t* _l;
        CUDA_CALL(hipMemcpyFromSymbol(&_l, HIP_SYMBOL(d_light), sizeof(hash64_t*)));
        *_light = _l;
    }
    if (_light_size) {
        uint32_t _ls;
        CUDA_CALL(hipMemcpyFromSymbol(&_ls, HIP_SYMBOL(d_light_size), sizeof(uint32_t)));
        *_light_size = _ls;
    }
}

void set_header(hash32_t _header) { CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_header), &_header, sizeof(hash32_t))); }

void set_target(uint64_t _target) { CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_target), &_target, sizeof(uint64_t))); }
